#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <vector>
#include "CudaLib.h"

#define byte unsigned char

namespace CudaLib
{
#if 0
}	// indent guard
#endif

__global__ void cudaDifference(int *a, int *b)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	b[i] = a[i] * 2;
	return;
}

__global__ void cudaDifference(int* a, int* b, int* c)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	c[i] = a[i] + b[i];
	return;
}

__global__ void cudaDifferenceBytes(int* a, int* b, int* c)
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int i = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	byte* aBytes = (byte*)a;
	byte* bBytes = (byte*)b;
	byte* cBytes = (byte*)c;

	cBytes[4 * i] = 128 + (aBytes[4 * i] - bBytes[4 * i]) / 2; //Blue
	cBytes[4 * i + 1] = 128 + (aBytes[4 * i + 1] - bBytes[4 * i + 1]) / 2; //Green
	cBytes[4 * i + 2] = 128 + (aBytes[4 * i + 2] - bBytes[4 * i + 2]) / 2; //Red
	cBytes[4 * i + 3] = 255; //Alpha

	return;
}

void computeDifference(int *a, int *b, int n)
{
	int byteCount = n * sizeof(int);
	int *cudaA;
	int *cudaB;

	hipMalloc(&cudaA, byteCount);
	hipMalloc(&cudaB, byteCount);

	hipMemcpy(cudaA, a, byteCount, hipMemcpyHostToDevice);

	dim3 blockSize = (1, 1);
	cudaDifference <<<blockSize, 1024>>> (cudaA, cudaB);
	hipDeviceSynchronize();

	hipMemcpy(b, cudaB, byteCount, hipMemcpyDeviceToHost);

	hipFree(cudaA);
	hipFree(cudaB);
}

void computeDifference2(int* a, int* b, int* c, int n)
{
	int byteCount = n * sizeof(int);
	int* cudaA;
	int* cudaB;
	int* cudaC;

	hipMalloc(&cudaA, byteCount);
	hipMalloc(&cudaB, byteCount);
	hipMalloc(&cudaC, byteCount);

	hipMemcpy(cudaA, a, byteCount, hipMemcpyHostToDevice);
	hipMemcpy(cudaB, b, byteCount, hipMemcpyHostToDevice);

	dim3 blockSize = (1, 1);
	cudaDifference << <blockSize, 1024 >> > (cudaA, cudaB, cudaC);
	hipDeviceSynchronize();

	hipMemcpy(c, cudaC, byteCount, hipMemcpyDeviceToHost);

	hipFree(cudaA);
	hipFree(cudaB);
	hipFree(cudaC);
}

void computeDifferenceBytes(int* a, int* b, int* c, int n)
{
	int byteCount = n * sizeof(int);
	int* cudaA;
	int* cudaB;
	int* cudaC;

	hipMalloc(&cudaA, byteCount);
	hipMalloc(&cudaB, byteCount);
	hipMalloc(&cudaC, byteCount);

	hipMemcpy(cudaA, a, byteCount, hipMemcpyHostToDevice);
	hipMemcpy(cudaB, b, byteCount, hipMemcpyHostToDevice);

	dim3 blockSize = (1, 384);//1,8100
	dim3 threadCount = 216; //256
	cudaDifferenceBytes << <blockSize, 256 >> > (cudaA, cudaB, cudaC);
	hipDeviceSynchronize();

	hipMemcpy(c, cudaC, byteCount, hipMemcpyDeviceToHost);

	hipFree(cudaA);
	hipFree(cudaB);
	hipFree(cudaC);
}
}
