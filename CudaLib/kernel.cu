#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <vector>
#include "CudaLib.h"

#define byte unsigned char

namespace CudaLib
{
#if 0
}	// indent guard
#endif

__global__ void cudaDifference(int *a, int *b)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	b[i] = a[i] * 2;
	return;
}

__global__ void cudaDifference(int* a, int* b, int* c)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	c[i] = a[i] + b[i];
	return;
}

__global__ void cudaDifferenceBytes(int* a, int* b, int* c)
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int i = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	int differenceThreshold = 30;

	byte* aBytes = (byte*)a;
	byte* bBytes = (byte*)b;
	byte* cBytes = (byte*)c;

	int byte1 = 4 * i;
	int byte2 = 4 * i + 1;
	int byte3 = 4 * i + 2;
	int byte4 = 4 * i + 3;

	cBytes[byte1] = 128;
	cBytes[byte2] = 128;
	cBytes[byte3] = 128;
	cBytes[byte4] = 255;

	if (abs((int)aBytes[byte1] - (int)bBytes[byte1]) > differenceThreshold)
		cBytes[byte1] += (aBytes[byte1] - bBytes[byte1]) / 2;

	if (abs((int)aBytes[byte2] - (int)bBytes[byte2]) > differenceThreshold)
		cBytes[byte2] += (aBytes[byte2] - bBytes[byte2]) / 2;

	if (abs((int)aBytes[byte3] - (int)bBytes[byte3]) > differenceThreshold)
		cBytes[byte3] += (aBytes[byte3] - bBytes[byte3]) / 2;

	return;
}

void computeDifference(int *a, int *b, int n)
{
	int byteCount = n * sizeof(int);
	int *cudaA;
	int *cudaB;

	hipMalloc(&cudaA, byteCount);
	hipMalloc(&cudaB, byteCount);

	hipMemcpy(cudaA, a, byteCount, hipMemcpyHostToDevice);

	dim3 blockSize = (1, 1);
	cudaDifference <<<blockSize, 1024>>> (cudaA, cudaB);
	hipDeviceSynchronize();

	hipMemcpy(b, cudaB, byteCount, hipMemcpyDeviceToHost);

	hipFree(cudaA);
	hipFree(cudaB);
}

void computeDifference2(int* a, int* b, int* c, int n)
{
	int byteCount = n * sizeof(int);
	int* cudaA;
	int* cudaB;
	int* cudaC;

	hipMalloc(&cudaA, byteCount);
	hipMalloc(&cudaB, byteCount);
	hipMalloc(&cudaC, byteCount);

	hipMemcpy(cudaA, a, byteCount, hipMemcpyHostToDevice);
	hipMemcpy(cudaB, b, byteCount, hipMemcpyHostToDevice);

	dim3 blockSize = (1, 1);
	cudaDifference << <blockSize, 1024 >> > (cudaA, cudaB, cudaC);
	hipDeviceSynchronize();

	hipMemcpy(c, cudaC, byteCount, hipMemcpyDeviceToHost);

	hipFree(cudaA);
	hipFree(cudaB);
	hipFree(cudaC);
}

void computeDifferenceBytes(int* a, int* b, int* c, int n)
{
	int byteCount = n * sizeof(int);
	int* cudaA;
	int* cudaB;
	int* cudaC;

	hipMalloc(&cudaA, byteCount);
	hipMalloc(&cudaB, byteCount);
	hipMalloc(&cudaC, byteCount);

	hipMemcpy(cudaA, a, byteCount, hipMemcpyHostToDevice);
	hipMemcpy(cudaB, b, byteCount, hipMemcpyHostToDevice);

	dim3 blockSize = (1, 384);//1,8100
	dim3 threadCount = 216; //256
	cudaDifferenceBytes << <blockSize, 256 >> > (cudaA, cudaB, cudaC);
	hipDeviceSynchronize();

	hipMemcpy(c, cudaC, byteCount, hipMemcpyDeviceToHost);

	hipFree(cudaA);
	hipFree(cudaB);
	hipFree(cudaC);
}
}
