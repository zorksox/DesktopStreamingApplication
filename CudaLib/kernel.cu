#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <vector>
#include "CudaLib.h"

#define byte unsigned char

namespace CudaLib
{
#if 0
}	// indent guard
#endif

__global__ void cudaDifference(int *a, int *b)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	b[i] = a[i] * 2;
	return;
}

__global__ void cudaDifference(int* a, int* b, int* c)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	c[i] = a[i] + b[i];
	return;
}

__global__ void cudaDifferenceBytes(int* a, int* b, int* c)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	byte* aBytes = (byte*)a;
	aBytes[0] = 200;
	aBytes[1] = 200;
	aBytes[2] = 200;
	aBytes[3] = 200;

	return;
}

void computeDifference(int *a, int *b, int n)
{
	int byteCount = n * sizeof(int);
	int *cudaA;
	int *cudaB;

	hipMalloc(&cudaA, byteCount);
	hipMalloc(&cudaB, byteCount);

	hipMemcpy(cudaA, a, byteCount, hipMemcpyHostToDevice);

	dim3 blockSize = (1, 1);
	cudaDifference <<<blockSize, 1024>>> (cudaA, cudaB);
	hipDeviceSynchronize();

	hipMemcpy(b, cudaB, byteCount, hipMemcpyDeviceToHost);

	hipFree(cudaA);
	hipFree(cudaB);
}

void computeDifference2(int* a, int* b, int* c, int n)
{
	int byteCount = n * sizeof(int);
	int* cudaA;
	int* cudaB;
	int* cudaC;

	hipMalloc(&cudaA, byteCount);
	hipMalloc(&cudaB, byteCount);
	hipMalloc(&cudaC, byteCount);

	hipMemcpy(cudaA, a, byteCount, hipMemcpyHostToDevice);
	hipMemcpy(cudaB, b, byteCount, hipMemcpyHostToDevice);

	dim3 blockSize = (1, 1);
	cudaDifference << <blockSize, 1024 >> > (cudaA, cudaB, cudaC);
	hipDeviceSynchronize();

	hipMemcpy(c, cudaC, byteCount, hipMemcpyDeviceToHost);

	hipFree(cudaA);
	hipFree(cudaB);
	hipFree(cudaC);
}

void computeDifferenceBytes(int* a, int* b, int* c, int n)
{
	int byteCount = n * sizeof(int);
	int* cudaA;
	int* cudaB;
	int* cudaC;

	hipMalloc(&cudaA, byteCount);
	hipMalloc(&cudaB, byteCount);
	hipMalloc(&cudaC, byteCount);

	hipMemcpy(cudaA, a, byteCount, hipMemcpyHostToDevice);
	hipMemcpy(cudaB, b, byteCount, hipMemcpyHostToDevice);

	dim3 blockSize = (1, 1);
	cudaDifferenceBytes << <blockSize, 1024 >> > (cudaA, cudaB, cudaC);
	hipDeviceSynchronize();

	hipMemcpy(a, cudaA, byteCount, hipMemcpyDeviceToHost);

	hipFree(cudaA);
	hipFree(cudaB);
	hipFree(cudaC);
}
}
