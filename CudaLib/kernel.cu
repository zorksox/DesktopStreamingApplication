#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "CudaLib.h"

namespace CudaLib
{
#if 0
}	// indent guard
#endif


void complexCalcOriginal(int *a, int *b, int n)
{
	for (int i = 0; i < n; i++) {
		b[i] = a[i] * 2;
	}
}


__global__ void complexCalcFastLoop(int *a, int *b, int n)
{
	int i = threadIdx.x;
	if (i < n) {
		b[i] = a[i] * 2;
	}
}

void complexCalcFast(int *a, int *b, int n)
{
	int *dIn;
	int *dOut;
	hipHostMalloc((void**)&dIn, n * sizeof(int));
	hipHostMalloc((void**)&dOut, n * sizeof(int));
	hipMemcpy(dIn, a, n * sizeof(int), hipMemcpyHostToDevice);

	complexCalcFastLoop <<<1, n>>> (dIn, dOut, n);
	hipDeviceSynchronize();

	hipMemcpy(b, dOut, n * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dIn);
	hipFree(dOut);
}

}
