#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <vector>
#include "CudaLib.h"

#define byte unsigned char

namespace CudaLib
{
#if 0
}	// indent guard
#endif

__global__ void cudaDifference(int *a, int *b)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	b[i] = a[i] * 2;
	return;
}

void computeDifference(int *a, int *b, int n)
{
	int byteCount = n * sizeof(int);
	int *cudaA;
	int *cudaB;
	hipMalloc(&cudaA, byteCount);
	hipMalloc(&cudaB, byteCount);
	hipMemcpy(cudaA, a, byteCount, hipMemcpyHostToDevice);

	dim3 blockSize = (1, 1);
	cudaDifference <<<blockSize, 1024>>> (cudaA, cudaB);
	hipDeviceSynchronize();

	hipMemcpy(b, cudaB, byteCount, hipMemcpyDeviceToHost);
	hipFree(cudaA);
	hipFree(cudaB);
}
}
